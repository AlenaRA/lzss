#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <string.h>
#include <limits.h>
#include <hip/hip_runtime.h>

#include "common.h"
#include "lzss_gpu_help.h"

#define DISP_BITS 11
#define LEN_BITS 4
#define WINDOW ((1<<DISP_BITS))
#define MAX_MATCH ((1<<LEN_BITS) + 1)
#define MIN_MATCH 2
#define min(x,y) (((x) < (y)) ? (x) : (y))
#define max(x,y) (((x) < (y)) ? (y) : (x))

__device__ uint16_t match_len(uint8_t* old, uint8_t* fwd, uint16_t fwd_max)
{
  uint16_t i;
  for(i=0;(i<min(fwd_max,MAX_MATCH)) && (old[i]==fwd[i]);i++);
  return i;
}

__device__ void window_match(uint8_t* word, uint16_t back_max, uint16_t fwd_max, match_expanded_t* dst)
{
  uint16_t best = 0;
  uint16_t offset = 0;

  for(uint16_t i=1;i<back_max;i++)
  {
    uint8_t* tmp = word - i;
    uint16_t curr = match_len(tmp,word,fwd_max+i);
    if(curr > best)
    {
      offset = i;
      best = curr;
    }
  }

  dst->d = offset;
  dst->l = best;
}

#define MATCH_TOP(i) ((i) >> LEN_BITS)
#define MATCH_BOT(i) ((i) & ((1<<(LEN_BITS))-1))

void unpack_match(match_expanded_t* expanded, match_t* match)
{
  uint16_t data = match->dl;
  expanded->d = MATCH_TOP(data) + 1;
  expanded->l = MATCH_BOT(data) + 2;
}

#define TO_TOP(i) ((i) << LEN_BITS)
#define TO_BOT(i) MATCH_BOT((i))
#define PACK(d,l) (TO_TOP((d)) | TO_BOT((l)))

__device__ void pack_match(match_t* match, match_expanded_t* expanded)
{
  match->dl = PACK(expanded->d-1,expanded->l-2);
}

#define PUT_BIT(bit,idx) ((bit) << (7-((idx)%8)))
#define IDX_BY_BIT(arr,idx) ((arr)[(idx)/8])
#define GET_BIT(arr,idx) ((IDX_BY_BIT(arr,idx) >> (7-((idx)%8))) & 0x1)

/* Make sure flags is zeroed out before passed in */
__global__ void gpu_compress(uint8_t* input, uint64_t input_len, uint8_t* dst, uint8_t* flags, comp_size_t *size)
{
  /* threadIdx.x; */
  uint64_t global_idx = blockIdx.x * blockDim.x + threadIdx.x;

  uint64_t partition_size = (global_idx * MAX_MATCH * 2 >= input_len) ?
    0 : min(input_len - global_idx * MAX_MATCH * 2, MAX_MATCH * 2);
  int64_t partition_off = (global_idx * MAX_MATCH * 2);

  input = input + partition_off;
  dst = dst + partition_off;
  flags = flags + partition_off;
  size = size + global_idx;

  match_expanded_t match;
  match_t m;
  uint64_t i=0;
  uint64_t w=0;
  uint64_t b=0;
  uint8_t *curr;
  /*
  printf("thread ID: %ld, last idx: %ld\n", global_idx + 100000, min((int64_t)(input_len-partition_off),(int64_t)partition_size));
  return;
  */
  for(;i<min((int64_t)(input_len-partition_off),(int64_t)partition_size) && partition_size > 0;)
  {
    curr = input + i;

    printf("OFFSET %5i\n",partition_off + i + 100000);

    uint64_t window_offset = min(partition_off + i,WINDOW);
    window_match(curr, window_offset, min((uint16_t) input_len-partition_off-i, (uint16_t) partition_size-i), &match);
    if( match.l < MIN_MATCH )
    {
      /* add 0 bit and the byte */
      for(int j=0;j<max(1,match.l);j++)
      {
        /* IDX_BY_BIT(flags,b+j) |= PUT_BIT(0,b+j); */
        flags[b+j] = 0;
        dst[w+j] = curr[j];
      }
      b += max(1,match.l);
      w += max(1,match.l);
      i += max(1,match.l);
    }
    else
    {
      /* match.d is displacement */
      /* match.l is length of match */
      /* IDX_BY_BIT(flags,b) |= PUT_BIT(1,b); */
      flags[b] = 1;
      pack_match(&m,&match);
      /* memcpy(dst + w,&m,sizeof(match_t)); */
      m = *((match_t*)(dst + w));
      i += match.l;
      w += sizeof(match_t);
      b++;
    }

  }

  size->b = b;
  size->w = w;
  printf("DONE %ld\n", global_idx + 1000000);
}

#define THREADS 256
#define BLOCKS ((input_len + (THREADS * MAX_MATCH * 2) + 1)/(THREADS * MAX_MATCH * 2))
#define PARTITION_SIZE ((input_len + THREADS * BLOCKS - 1)/(THREADS * BLOCKS))
comp_size_t compress(uint8_t* input, uint64_t input_len, uint8_t* dst, uint8_t* flags)
{
  uint8_t *gpu_input, *gpu_dst, *gpu_flags;
  comp_size_t *sizes, *gpu_sizes;
  printf("length: %ld, THREADS: %d, BLOCKS: %ld\n",input_len, THREADS, BLOCKS); fflush(0);
  hipDeviceSynchronize();

  hipMalloc(&gpu_input, input_len);
  hipMalloc(&gpu_dst, input_len);
  hipMalloc(&gpu_flags, input_len);
  hipMalloc(&gpu_sizes, BLOCKS * THREADS * sizeof(comp_size_t));

  hipMemcpy(gpu_input,input,input_len,hipMemcpyHostToDevice);

  hipDeviceSynchronize();

  gpu_compress <<<BLOCKS,THREADS>>> (gpu_input, input_len, gpu_dst, gpu_flags, gpu_sizes);

  hipDeviceSynchronize();

  comp_size_t cmp_size;
  cmp_size.b = 0;
  cmp_size.w = 0;

  sizes = (comp_size_t*) malloc(THREADS * BLOCKS * sizeof(comp_size_t));
  hipMemcpy(sizes, gpu_sizes, THREADS * BLOCKS * sizeof(comp_size_t), hipMemcpyDeviceToHost);

  uint8_t *flag_buf = (uint8_t*) malloc(input_len);
  for(int i=0;i<THREADS*BLOCKS;i++)
  {
    hipMemcpy(dst + cmp_size.w , gpu_dst + i * PARTITION_SIZE, sizes[i].w, hipMemcpyDeviceToHost);
    hipMemcpy(flag_buf + cmp_size.b , gpu_flags + i * PARTITION_SIZE, sizes[i].b, hipMemcpyDeviceToHost);
    fprintf(stdout,"bsize[%3.3d] = %3.3d, wsize[%3.3d] = %3.3d\n",i,sizes[i].b,i,sizes[i].w);
    cmp_size.b += sizes[i].b;
    cmp_size.w += sizes[i].w;
  }

  fprintf(stdout,"flag bits: %ld, stuff bytes: %ld\n",cmp_size.b,cmp_size.w);

  memset(flags,0,(cmp_size.b+7)/8);
  for(int i=0;i<cmp_size.b;i++)
  {
    IDX_BY_BIT(flags,i) |= PUT_BIT(flag_buf[i],i);
  }

  return cmp_size;
}

uint64_t decompress(uint8_t* input, uint8_t* flags, uint64_t input_len, uint8_t* dst)
{
  uint64_t b=0;
  uint64_t i=0;
  uint64_t w=0;

  register uint8_t *curr;
  register uint8_t bit;
  match_t *match;
  match_expanded_t m;

  for(;i<input_len;)
  {
    curr = input + i;
    bit = GET_BIT(flags,b);
    
    if(!bit)
    {
      dst[w] = input[i];
      w++; i++; b++;
    }
    else
    {
      match = (match_t*) curr;
      unpack_match(&m,match);
      for(int j=0;j<m.l;j++)
      {
        dst[w + j] = dst[w + j - m.d];
      }
      w += m.l;
      i += sizeof(match_t);
      b++;
    }
  }

  return w;
}


compressed_t *lzss_compress(decomp_t *decomp)
{
  uint64_t len = decomp->content_len;
  uint8_t *flag = (uint8_t*) calloc(BITS_TO_CHARS(len), sizeof(uint8_t));
  if(!flag)
  {
    return NULL;
  }
  uint8_t *buf = (uint8_t*) malloc(len * sizeof(uint8_t));
  if(!buf)
  {
    free(flag);
    return NULL;
  }

  comp_size_t comp_sizes = compress(decomp->content,len,buf,flag);
  uint64_t comp_len = comp_sizes.w;
  uint64_t flag_bits = comp_sizes.b;
  uint64_t flag_bytes = BITS_TO_CHARS(flag_bits);

  compressed_t *comp = (compressed_t*) malloc(sizeof(compressed_t) + (flag_bytes + comp_len) * sizeof(uint8_t));
  if(!comp){
    free(flag);
    free(buf);
    return NULL;
  }
  comp->file_len = len;
  comp->content_len = comp_len;
  comp->flag_bits = flag_bits;
  memcpy(comp->content,flag,flag_bytes);
  memcpy(comp->content + flag_bytes,buf,comp->content_len);

  free(flag);
  free(buf);
  return comp;
}


decomp_t *lzss_decomp(compressed_t *comp)
{
  uint64_t file_len = comp->file_len;
  uint64_t comp_len = comp->content_len;
  uint64_t flag_bits = comp->flag_bits;
  uint64_t flag_bytes = BITS_TO_CHARS(flag_bits);

  uint8_t *flag_buf = comp->content;
  uint8_t *comp_buf = comp->content + flag_bytes;

  decomp_t *decomp = (decomp_t*) malloc(sizeof(decomp_t) + (file_len)*sizeof(uint8_t));
  if(!decomp)
  {
    return NULL;
  }

  uint64_t decomp_len = decompress(comp_buf,flag_buf,comp_len,decomp->content);
  decomp->content_len = decomp_len;

  return decomp;
}

